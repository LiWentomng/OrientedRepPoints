#include "hip/hip_runtime.h"
// Modified from https://github.com/pytorch/pytorch/blob/master/aten/src/ATen/native/cuda/GridSampler.cu

#include <ATen/ATen.h>
#include "grid_sampler_cuda.cuh"
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <c10/macros/Macros.h>

namespace mmdetection {

using namespace at::cuda::detail;

using mmdetection::detail::GridSamplerInterpolation;
using mmdetection::detail::GridSamplerPadding;

namespace {
  template <typename scalar_t>
  C10_LAUNCH_BOUNDS_1(1024)
  __global__ void grid_sampler_2d_forward_kernel_cuda(
      const int nthreads,
      TensorInfo<scalar_t, int> input,
      TensorInfo<scalar_t, int> grid,
      TensorInfo<scalar_t, int> output,
      const GridSamplerInterpolation interpolation_mode,
      const GridSamplerPadding padding_mode,
      bool align_corners) {

    int C = input.sizes[1];
    int inp_H = input.sizes[2];
    int inp_W = input.sizes[3];
    int out_H = grid.sizes[1];
    int out_W = grid.sizes[2];
    int inp_sN = input.strides[0];
    int inp_sC = input.strides[1];
    int inp_sH = input.strides[2];
    int inp_sW = input.strides[3];
    int grid_sN = grid.strides[0];
    int grid_sH = grid.strides[1];
    int grid_sW = grid.strides[2];
    int grid_sCoor = grid.strides[3];
    int out_sN = output.strides[0];
    int out_sC = output.strides[1];
    int out_sH = output.strides[2];
    int out_sW = output.strides[3];

    CUDA_KERNEL_LOOP(index, nthreads) {
      const int w = index % out_W;
      const int h = (index / out_W) % out_H;
      const int n = index / (out_H * out_W);
      const int grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

      // get the corresponding input x, y co-ordinates from grid
      scalar_t ix = grid.data[grid_offset];
      scalar_t iy = grid.data[grid_offset + grid_sCoor];

      ix = grid_sampler_compute_source_index(ix, inp_W, padding_mode, align_corners);
      iy = grid_sampler_compute_source_index(iy, inp_H, padding_mode, align_corners);

      if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
        // get NE, NW, SE, SW pixel values from (x, y)
        int ix_nw = static_cast<int>(::floor(ix));
        int iy_nw = static_cast<int>(::floor(iy));
        int ix_ne = ix_nw + 1;
        int iy_ne = iy_nw;
        int ix_sw = ix_nw;
        int iy_sw = iy_nw + 1;
        int ix_se = ix_nw + 1;
        int iy_se = iy_nw + 1;

        // get surfaces to each neighbor:
        scalar_t nw = (ix_se - ix)    * (iy_se - iy);
        scalar_t ne = (ix    - ix_sw) * (iy_sw - iy);
        scalar_t sw = (ix_ne - ix)    * (iy    - iy_ne);
        scalar_t se = (ix    - ix_nw) * (iy    - iy_nw);

        // calculate bilinear weighted pixel value and set output pixel
        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCHW = output.data + n * out_sN + h * out_sH + w * out_sW;
        for (int c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
          *out_ptr_NCHW = static_cast<scalar_t>(0);
          if (within_bounds_2d(iy_nw, ix_nw, inp_H, inp_W)) {
            *out_ptr_NCHW += inp_ptr_NC[iy_nw * inp_sH + ix_nw * inp_sW] * nw;
          }
          if (within_bounds_2d(iy_ne, ix_ne, inp_H, inp_W)) {
            *out_ptr_NCHW += inp_ptr_NC[iy_ne * inp_sH + ix_ne * inp_sW] * ne;
          }
          if (within_bounds_2d(iy_sw, ix_sw, inp_H, inp_W)) {
            *out_ptr_NCHW += inp_ptr_NC[iy_sw * inp_sH + ix_sw * inp_sW] * sw;
          }
          if (within_bounds_2d(iy_se, ix_se, inp_H, inp_W)) {
            *out_ptr_NCHW += inp_ptr_NC[iy_se * inp_sH + ix_se * inp_sW] * se;
          }
        }
      } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
        int ix_nearest = static_cast<int>(::round(ix));
        int iy_nearest = static_cast<int>(::round(iy));

        // assign nearest neighor pixel value to output pixel
        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCHW = output.data + n * out_sN + h * out_sH + w * out_sW;
        for (int c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
          if (within_bounds_2d(iy_nearest, ix_nearest, inp_H, inp_W)) {
            *out_ptr_NCHW = inp_ptr_NC[iy_nearest * inp_sH + ix_nearest * inp_sW];
          } else {
            *out_ptr_NCHW = static_cast<scalar_t>(0);
          }
        }
      }
    }
  }

  template <typename scalar_t>
  C10_LAUNCH_BOUNDS_1(1024)
  __global__ void grid_sampler_3d_forward_kernel_cuda(
      const int nthreads,
      TensorInfo<scalar_t, int> input,
      TensorInfo<scalar_t, int> grid,
      TensorInfo<scalar_t, int> output,
      const GridSamplerInterpolation interpolation_mode,
      const GridSamplerPadding padding_mode,
      bool align_corners) {

    int C = input.sizes[1];
    int inp_D = input.sizes[2];
    int inp_H = input.sizes[3];
    int inp_W = input.sizes[4];
    int out_D = grid.sizes[1];
    int out_H = grid.sizes[2];
    int out_W = grid.sizes[3];
    int inp_sN = input.strides[0];
    int inp_sC = input.strides[1];
    int inp_sD = input.strides[2];
    int inp_sH = input.strides[3];
    int inp_sW = input.strides[4];
    int grid_sN = grid.strides[0];
    int grid_sD = grid.strides[1];
    int grid_sH = grid.strides[2];
    int grid_sW = grid.strides[3];
    int grid_sCoor = grid.strides[4];
    int out_sN = output.strides[0];
    int out_sC = output.strides[1];
    int out_sD = output.strides[2];
    int out_sH = output.strides[3];
    int out_sW = output.strides[4];

    CUDA_KERNEL_LOOP(index, nthreads) {
      const int w = index % out_W;
      const int h = (index / out_W) % out_H;
      const int d = (index / (out_H * out_W)) % out_D;
      const int n = index / (out_D * out_H * out_W);
      const int grid_offset = n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;

      // get the corresponding input x, y, z co-ordinates from grid
      scalar_t ix = grid.data[grid_offset];
      scalar_t iy = grid.data[grid_offset + grid_sCoor];
      scalar_t iz = grid.data[grid_offset + 2 * grid_sCoor];

      ix = grid_sampler_compute_source_index(ix, inp_W, padding_mode, align_corners);
      iy = grid_sampler_compute_source_index(iy, inp_H, padding_mode, align_corners);
      iz = grid_sampler_compute_source_index(iz, inp_D, padding_mode, align_corners);

      if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
        // get corner pixel values from (x, y, z)
        // for 4d, we used north-east-south-west
        // for 5d, we add top-bottom
        int ix_tnw = static_cast<int>(::floor(ix));
        int iy_tnw = static_cast<int>(::floor(iy));
        int iz_tnw = static_cast<int>(::floor(iz));

        int ix_tne = ix_tnw + 1;
        int iy_tne = iy_tnw;
        int iz_tne = iz_tnw;

        int ix_tsw = ix_tnw;
        int iy_tsw = iy_tnw + 1;
        int iz_tsw = iz_tnw;

        int ix_tse = ix_tnw + 1;
        int iy_tse = iy_tnw + 1;
        int iz_tse = iz_tnw;

        int ix_bnw = ix_tnw;
        int iy_bnw = iy_tnw;
        int iz_bnw = iz_tnw + 1;

        int ix_bne = ix_tnw + 1;
        int iy_bne = iy_tnw;
        int iz_bne = iz_tnw + 1;

        int ix_bsw = ix_tnw;
        int iy_bsw = iy_tnw + 1;
        int iz_bsw = iz_tnw + 1;

        int ix_bse = ix_tnw + 1;
        int iy_bse = iy_tnw + 1;
        int iz_bse = iz_tnw + 1;

        // get surfaces to each neighbor:
        scalar_t tnw = (ix_bse - ix)    * (iy_bse - iy)    * (iz_bse - iz);
        scalar_t tne = (ix    - ix_bsw) * (iy_bsw - iy)    * (iz_bsw - iz);
        scalar_t tsw = (ix_bne - ix)    * (iy    - iy_bne) * (iz_bne - iz);
        scalar_t tse = (ix    - ix_bnw) * (iy    - iy_bnw) * (iz_bnw - iz);
        scalar_t bnw = (ix_tse - ix)    * (iy_tse - iy)    * (iz - iz_tse);
        scalar_t bne = (ix    - ix_tsw) * (iy_tsw - iy)    * (iz - iz_tsw);
        scalar_t bsw = (ix_tne - ix)    * (iy    - iy_tne) * (iz - iz_tne);
        scalar_t bse = (ix    - ix_tnw) * (iy    - iy_tnw) * (iz - iz_tnw);

        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCDHW = output.data + n * out_sN + d * out_sD + h * out_sH + w * out_sW;
        for (int c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
          //   (c, iz_tnw, iy_tnw, ix_tnw) * tnw + (c, iz_tne, iy_tne, ix_tne) * tne
          // + (c, iz_tsw, iy_tsw, ix_tsw) * tsw + (c, iz_tse, iy_tse, ix_tse) * tse
          // + (c, iz_bnw, iy_bnw, ix_bnw) * bnw + (c, iz_bne, iy_bne, ix_bne) * bne
          // + (c, iz_bsw, iy_bsw, ix_bsw) * bsw + (c, iz_bse, iy_bse, ix_bse) * bse
          *out_ptr_NCDHW = static_cast<scalar_t>(0);
          if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_tnw * inp_sD + iy_tnw * inp_sH + ix_tnw * inp_sW] * tnw;
          }
          if (within_bounds_3d(iz_tne, iy_tne, ix_tne, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_tne * inp_sD + iy_tne * inp_sH + ix_tne * inp_sW] * tne;
          }
          if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_tsw * inp_sD + iy_tsw * inp_sH + ix_tsw * inp_sW] * tsw;
          }
          if (within_bounds_3d(iz_tse, iy_tse, ix_tse, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_tse * inp_sD + iy_tse * inp_sH + ix_tse * inp_sW] * tse;
          }
          if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_bnw * inp_sD + iy_bnw * inp_sH + ix_bnw * inp_sW] * bnw;
          }
          if (within_bounds_3d(iz_bne, iy_bne, ix_bne, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_bne * inp_sD + iy_bne * inp_sH + ix_bne * inp_sW] * bne;
          }
          if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_bsw * inp_sD + iy_bsw * inp_sH + ix_bsw * inp_sW] * bsw;
          }
          if (within_bounds_3d(iz_bse, iy_bse, ix_bse, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_bse * inp_sD + iy_bse * inp_sH + ix_bse * inp_sW] * bse;
          }
        }
      } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
        int ix_nearest = static_cast<int>(::round(ix));
        int iy_nearest = static_cast<int>(::round(iy));
        int iz_nearest = static_cast<int>(::round(iz));

        // assign nearest neighor pixel value to output pixel
        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCDHW = output.data + n * out_sN + d * out_sD + h * out_sH + w * out_sW;
        for (int c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
          if (within_bounds_3d(iz_nearest, iy_nearest, ix_nearest, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW = inp_ptr_NC[iz_nearest * inp_sD + iy_nearest * inp_sH + ix_nearest * inp_sW];
          } else {
            *out_ptr_NCDHW = static_cast<scalar_t>(0);
          }
        }
      }
    }
  }

  template <typename scalar_t>
  C10_LAUNCH_BOUNDS_1(1024)
  __global__ void grid_sampler_2d_backward_kernel_cuda(
      const int nthreads,
      TensorInfo<scalar_t, int> grad_output,
      TensorInfo<scalar_t, int> input,
      TensorInfo<scalar_t, int> grid,
      TensorInfo<scalar_t, int> grad_input,  // initialized to zeros
      TensorInfo<scalar_t, int> grad_grid,   // initialized to empty
      const GridSamplerInterpolation interpolation_mode,
      const GridSamplerPadding padding_mode,
      bool align_corners) {

    int C = input.sizes[1];
    int inp_H = input.sizes[2];
    int inp_W = input.sizes[3];
    int out_H = grid.sizes[1];
    int out_W = grid.sizes[2];
    int inp_sN = input.strides[0];
    int inp_sC = input.strides[1];
    int inp_sH = input.strides[2];
    int inp_sW = input.strides[3];
    int grid_sN = grid.strides[0];
    int grid_sH = grid.strides[1];
    int grid_sW = grid.strides[2];
    int grid_sCoor = grid.strides[3];
    int gOut_sN = grad_output.strides[0];
    int gOut_sC = grad_output.strides[1];
    int gOut_sH = grad_output.strides[2];
    int gOut_sW = grad_output.strides[3];
    int gInp_sN = grad_input.strides[0];
    int gInp_sC = grad_input.strides[1];
    int gInp_sH = grad_input.strides[2];
    int gInp_sW = grad_input.strides[3];
    int gGrid_sW = grad_grid.strides[2];

    CUDA_KERNEL_LOOP(index, nthreads) {
      const int w = index % out_W;
      const int h = (index / out_W) % out_H;
      const int n = index / (out_H * out_W);
      const int grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

      // get the corresponding input x, y co-ordinates from grid
      scalar_t ix = grid.data[grid_offset];
      scalar_t iy = grid.data[grid_offset + grid_sCoor];

      // multipliers for gradients on ix and iy
      scalar_t gix_mult, giy_mult;
      ix = grid_sampler_compute_source_index_set_grad(ix, inp_W, padding_mode, align_corners, &gix_mult);
      iy = grid_sampler_compute_source_index_set_grad(iy, inp_H, padding_mode, align_corners, &giy_mult);

      if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
        // get NE, NW, SE, SW pixel values from (x, y)
        int ix_nw = static_cast<int>(::floor(ix));
        int iy_nw = static_cast<int>(::floor(iy));
        int ix_ne = ix_nw + 1;
        int iy_ne = iy_nw;
        int ix_sw = ix_nw;
        int iy_sw = iy_nw + 1;
        int ix_se = ix_nw + 1;
        int iy_se = iy_nw + 1;

        // get surfaces to each neighbor:
        scalar_t nw = (ix_se - ix)    * (iy_se - iy);
        scalar_t ne = (ix    - ix_sw) * (iy_sw - iy);
        scalar_t sw = (ix_ne - ix)    * (iy    - iy_ne);
        scalar_t se = (ix    - ix_nw) * (iy    - iy_nw);

        scalar_t gix = static_cast<scalar_t>(0), giy = static_cast<scalar_t>(0);
        scalar_t *gOut_ptr_NCHW = grad_output.data + n * gOut_sN + h * gOut_sH + w * gOut_sW;
        scalar_t *gInp_ptr_NC = grad_input.data + n * gInp_sN;
        scalar_t *inp_ptr_NC = input.data + n * inp_sN;
        for (int c = 0; c < C; ++c, inp_ptr_NC += inp_sC, gInp_ptr_NC += gInp_sC, gOut_ptr_NCHW += gOut_sC) {
          scalar_t gOut = *gOut_ptr_NCHW;

          // calculate and set grad_input
          safe_add_2d(gInp_ptr_NC, iy_nw, ix_nw, gInp_sH, gInp_sW, inp_H, inp_W, nw * gOut);
          safe_add_2d(gInp_ptr_NC, iy_ne, ix_ne, gInp_sH, gInp_sW, inp_H, inp_W, ne * gOut);
          safe_add_2d(gInp_ptr_NC, iy_sw, ix_sw, gInp_sH, gInp_sW, inp_H, inp_W, sw * gOut);
          safe_add_2d(gInp_ptr_NC, iy_se, ix_se, gInp_sH, gInp_sW, inp_H, inp_W, se * gOut);

          // calculate grad_grid
          if (within_bounds_2d(iy_nw, ix_nw, inp_H, inp_W)) {
            scalar_t nw_val = inp_ptr_NC[iy_nw * inp_sH + ix_nw * inp_sW];
            gix -= nw_val * (iy_se - iy) * gOut;
            giy -= nw_val * (ix_se - ix) * gOut;
          }
          if (within_bounds_2d(iy_ne, ix_ne, inp_H, inp_W)) {
            scalar_t ne_val = inp_ptr_NC[iy_ne * inp_sH + ix_ne * inp_sW];
            gix += ne_val * (iy_sw - iy) * gOut;
            giy -= ne_val * (ix - ix_sw) * gOut;
          }
          if (within_bounds_2d(iy_sw, ix_sw, inp_H, inp_W)) {
            scalar_t sw_val = inp_ptr_NC[iy_sw * inp_sH + ix_sw * inp_sW];
            gix -= sw_val * (iy - iy_ne) * gOut;
            giy += sw_val * (ix_ne - ix) * gOut;
          }
          if (within_bounds_2d(iy_se, ix_se, inp_H, inp_W)) {
            scalar_t se_val = inp_ptr_NC[iy_se * inp_sH + ix_se * inp_sW];
            gix += se_val * (iy - iy_nw) * gOut;
            giy += se_val * (ix - ix_nw) * gOut;
          }
        }

        // assuming grad_grid is contiguous
        // thus we can
        //   1. use index with gGrid_sW to directly compute gGrid_ptr_NHW
        //   2. directly assign to gGrid_ptr_NHW[0], gGrid_ptr_NHW[1]
        scalar_t *gGrid_ptr_NHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NHW[0] = gix_mult * gix;
        gGrid_ptr_NHW[1] = giy_mult * giy;
      } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
        int ix_nearest = static_cast<int>(::round(ix));
        int iy_nearest = static_cast<int>(::round(iy));

        // assign nearest neighor pixel value to output pixel
        scalar_t *gOut_ptr_NCHW = grad_output.data + n * gOut_sN + h * gOut_sH + w * gOut_sW;
        scalar_t *gInp_ptr_NC = grad_input.data + n * gInp_sN;
        for (int c = 0; c < C; ++c, gInp_ptr_NC += gInp_sC, gOut_ptr_NCHW += gOut_sC) {
          // calculate and set grad_input
          safe_add_2d(gInp_ptr_NC, iy_nearest, ix_nearest, gInp_sH, gInp_sW, inp_H, inp_W, *gOut_ptr_NCHW);
        }

        // assuming grad_grid is contiguous
        // thus we can
        //   1. use index with gGrid_sW to directly compute gGrid_ptr_NHW
        //   2. directly assign to gGrid_ptr_NHW[0], gGrid_ptr_NHW[1]
        scalar_t *gGrid_ptr_NHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NHW[0] = static_cast<scalar_t>(0);
        gGrid_ptr_NHW[1] = static_cast<scalar_t>(0);
      }
    }
  }

  template <typename scalar_t>
  C10_LAUNCH_BOUNDS_1(1024)
  __global__ void grid_sampler_3d_backward_kernel_cuda(
      const int nthreads,
      TensorInfo<scalar_t, int> grad_output,
      TensorInfo<scalar_t, int> input,
      TensorInfo<scalar_t, int> grid,
      TensorInfo<scalar_t, int> grad_input,  // initialized to zeros
      TensorInfo<scalar_t, int> grad_grid,   // initialized to empty
      const GridSamplerInterpolation interpolation_mode,
      const GridSamplerPadding padding_mode,
      bool align_corners) {

    int C = input.sizes[1];
    int inp_D = input.sizes[2];
    int inp_H = input.sizes[3];
    int inp_W = input.sizes[4];
    int out_D = grid.sizes[1];
    int out_H = grid.sizes[2];
    int out_W = grid.sizes[3];
    int inp_sN = input.strides[0];
    int inp_sC = input.strides[1];
    int inp_sD = input.strides[2];
    int inp_sH = input.strides[3];
    int inp_sW = input.strides[4];
    int grid_sN = grid.strides[0];
    int grid_sD = grid.strides[1];
    int grid_sH = grid.strides[2];
    int grid_sW = grid.strides[3];
    int grid_sCoor = grid.strides[4];
    int gOut_sN = grad_output.strides[0];
    int gOut_sC = grad_output.strides[1];
    int gOut_sD = grad_output.strides[2];
    int gOut_sH = grad_output.strides[3];
    int gOut_sW = grad_output.strides[4];
    int gInp_sN = grad_input.strides[0];
    int gInp_sC = grad_input.strides[1];
    int gInp_sD = grad_input.strides[2];
    int gInp_sH = grad_input.strides[3];
    int gInp_sW = grad_input.strides[4];
    int gGrid_sW = grad_grid.strides[3];

    CUDA_KERNEL_LOOP(index, nthreads) {
      const int w = index % out_W;
      const int h = (index / out_W) % out_H;
      const int d = (index / (out_H * out_W)) % out_D;
      const int n = index / (out_D * out_H * out_W);
      const int grid_offset = n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;

      // get the corresponding input x, y, z co-ordinates from grid
      scalar_t ix = grid.data[grid_offset];
      scalar_t iy = grid.data[grid_offset + grid_sCoor];
      scalar_t iz = grid.data[grid_offset + 2 * grid_sCoor];

      // multipliers for gradients on ix, iy, and iz
      scalar_t gix_mult, giy_mult, giz_mult;
      ix = grid_sampler_compute_source_index_set_grad(ix, inp_W, padding_mode, align_corners, &gix_mult);
      iy = grid_sampler_compute_source_index_set_grad(iy, inp_H, padding_mode, align_corners, &giy_mult);
      iz = grid_sampler_compute_source_index_set_grad(iz, inp_D, padding_mode, align_corners, &giz_mult);

      if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
        // get corner pixel values from (x, y, z)
        // for 4d, we used north-east-south-west
        // for 5d, we add top-bottom
        int ix_tnw = static_cast<int>(::floor(ix));
        int iy_tnw = static_cast<int>(::floor(iy));
        int iz_tnw = static_cast<int>(::floor(iz));

        int ix_tne = ix_tnw + 1;
        int iy_tne = iy_tnw;
        int iz_tne = iz_tnw;

        int ix_tsw = ix_tnw;
        int iy_tsw = iy_tnw + 1;
        int iz_tsw = iz_tnw;

        int ix_tse = ix_tnw + 1;
        int iy_tse = iy_tnw + 1;
        int iz_tse = iz_tnw;

        int ix_bnw = ix_tnw;
        int iy_bnw = iy_tnw;
        int iz_bnw = iz_tnw + 1;

        int ix_bne = ix_tnw + 1;
        int iy_bne = iy_tnw;
        int iz_bne = iz_tnw + 1;

        int ix_bsw = ix_tnw;
        int iy_bsw = iy_tnw + 1;
        int iz_bsw = iz_tnw + 1;

        int ix_bse = ix_tnw + 1;
        int iy_bse = iy_tnw + 1;
        int iz_bse = iz_tnw + 1;

        // get surfaces to each neighbor:
        scalar_t tnw = (ix_bse - ix)    * (iy_bse - iy)    * (iz_bse - iz);
        scalar_t tne = (ix    - ix_bsw) * (iy_bsw - iy)    * (iz_bsw - iz);
        scalar_t tsw = (ix_bne - ix)    * (iy    - iy_bne) * (iz_bne - iz);
        scalar_t tse = (ix    - ix_bnw) * (iy    - iy_bnw) * (iz_bnw - iz);
        scalar_t bnw = (ix_tse - ix)    * (iy_tse - iy)    * (iz - iz_tse);
        scalar_t bne = (ix    - ix_tsw) * (iy_tsw - iy)    * (iz - iz_tsw);
        scalar_t bsw = (ix_tne - ix)    * (iy    - iy_tne) * (iz - iz_tne);
        scalar_t bse = (ix    - ix_tnw) * (iy    - iy_tnw) * (iz - iz_tnw);

        scalar_t gix = static_cast<scalar_t>(0), giy = static_cast<scalar_t>(0), giz = static_cast<scalar_t>(0);
        scalar_t *gOut_ptr_NCDHW = grad_output.data + n * gOut_sN + d * gOut_sD + h * gOut_sH + w * gOut_sW;
        scalar_t *gInp_ptr_NC = grad_input.data + n * gInp_sN;
        scalar_t *inp_ptr_NC = input.data + n * inp_sN;
        // calculate bilinear weighted pixel value and set output pixel
        for (int c = 0; c < C; ++c, gOut_ptr_NCDHW += gOut_sC, gInp_ptr_NC += gInp_sC, inp_ptr_NC += inp_sC) {
          scalar_t gOut = *gOut_ptr_NCDHW;

          // calculate and set grad_input
          safe_add_3d(gInp_ptr_NC, iz_tnw, iy_tnw, ix_tnw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tnw * gOut);
          safe_add_3d(gInp_ptr_NC, iz_tne, iy_tne, ix_tne, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tne * gOut);
          safe_add_3d(gInp_ptr_NC, iz_tsw, iy_tsw, ix_tsw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tsw * gOut);
          safe_add_3d(gInp_ptr_NC, iz_tse, iy_tse, ix_tse, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tse * gOut);
          safe_add_3d(gInp_ptr_NC, iz_bnw, iy_bnw, ix_bnw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bnw * gOut);
          safe_add_3d(gInp_ptr_NC, iz_bne, iy_bne, ix_bne, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bne * gOut);
          safe_add_3d(gInp_ptr_NC, iz_bsw, iy_bsw, ix_bsw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bsw * gOut);
          safe_add_3d(gInp_ptr_NC, iz_bse, iy_bse, ix_bse, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bse * gOut);

          // calculate grad_grid
          if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, inp_D, inp_H, inp_W)) {
            scalar_t tnw_val = inp_ptr_NC[iz_tnw * inp_sD + iy_tnw * inp_sH + ix_tnw * inp_sW];
            gix -= tnw_val * (iy_bse - iy)    * (iz_bse - iz)    * gOut;
            giy -= tnw_val * (ix_bse - ix)    * (iz_bse - iz)    * gOut;
            giz -= tnw_val * (ix_bse - ix)    * (iy_bse - iy)    * gOut;
          }
          if (within_bounds_3d(iz_tne, iy_tne, ix_tne, inp_D, inp_H, inp_W)) {
            scalar_t tne_val = inp_ptr_NC[iz_tne * inp_sD + iy_tne * inp_sH + ix_tne * inp_sW];
            gix += tne_val * (iy_bsw - iy)    * (iz_bsw - iz)    * gOut;
            giy -= tne_val * (ix    - ix_bsw) * (iz_bsw - iz)    * gOut;
            giz -= tne_val * (ix    - ix_bsw) * (iy_bsw - iy)    * gOut;
          }
          if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, inp_D, inp_H, inp_W)) {
            scalar_t tsw_val = inp_ptr_NC[iz_tsw * inp_sD + iy_tsw * inp_sH + ix_tsw * inp_sW];
            gix -= tsw_val * (iy - iy_bne)    * (iz_bne - iz)    * gOut;
            giy += tsw_val * (ix_bne - ix)    * (iz_bne - iz)    * gOut;
            giz -= tsw_val * (ix_bne - ix)    * (iy    - iy_bne) * gOut;
          }
          if (within_bounds_3d(iz_tse, iy_tse, ix_tse, inp_D, inp_H, inp_W)) {
            scalar_t tse_val = inp_ptr_NC[iz_tse * inp_sD + iy_tse * inp_sH + ix_tse * inp_sW];
            gix += tse_val * (iy - iy_bnw)    * (iz_bnw - iz)    * gOut;
            giy += tse_val * (ix    - ix_bnw) * (iz_bnw - iz)    * gOut;
            giz -= tse_val * (ix    - ix_bnw) * (iy    - iy_bnw) * gOut;
          }
          if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, inp_D, inp_H, inp_W)) {
            scalar_t bnw_val = inp_ptr_NC[iz_bnw * inp_sD + iy_bnw * inp_sH + ix_bnw * inp_sW];
            gix -= bnw_val * (iy_tse - iy)    * (iz - iz_tse)    * gOut;
            giy -= bnw_val * (ix_tse - ix)    * (iz - iz_tse)    * gOut;
            giz += bnw_val * (ix_tse - ix)    * (iy_tse - iy)    * gOut;
          }
          if (within_bounds_3d(iz_bne, iy_bne, ix_bne, inp_D, inp_H, inp_W)) {
            scalar_t bne_val = inp_ptr_NC[iz_bne * inp_sD + iy_bne * inp_sH + ix_bne * inp_sW];
            gix += bne_val * (iy_tsw - iy)    * (iz - iz_tsw)    * gOut;
            giy -= bne_val * (ix    - ix_tsw) * (iz - iz_tsw)    * gOut;
            giz += bne_val * (ix    - ix_tsw) * (iy_tsw - iy)    * gOut;
          }
          if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, inp_D, inp_H, inp_W)) {
            scalar_t bsw_val = inp_ptr_NC[iz_bsw * inp_sD + iy_bsw * inp_sH + ix_bsw * inp_sW];
            gix -= bsw_val * (iy - iy_tne)    * (iz - iz_tne)    * gOut;
            giy += bsw_val * (ix_tne - ix)    * (iz - iz_tne)    * gOut;
            giz += bsw_val * (ix_tne - ix)    * (iy    - iy_tne) * gOut;
          }
          if (within_bounds_3d(iz_bse, iy_bse, ix_bse, inp_D, inp_H, inp_W)) {
            scalar_t bse_val = inp_ptr_NC[iz_bse * inp_sD + iy_bse * inp_sH + ix_bse * inp_sW];
            gix += bse_val * (iy - iy_tnw)    * (iz - iz_tnw)    * gOut;
            giy += bse_val * (ix    - ix_tnw) * (iz - iz_tnw)    * gOut;
            giz += bse_val * (ix    - ix_tnw) * (iy    - iy_tnw) * gOut;
          }
        }

        // assuming grad_grid is contiguous
        // thus we can
        //   1. use index with gGrid_sW to directly compute gGrid_ptr_NDHW
        //   2. directly assign to gGrid_ptr_NDHW[0], gGrid_ptr_NDHW[1], gGrid_ptr_NDHW[2]
        scalar_t *gGrid_ptr_NDHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NDHW[0] = gix_mult * gix;
        gGrid_ptr_NDHW[1] = giy_mult * giy;
        gGrid_ptr_NDHW[2] = giz_mult * giz;
      } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
        int ix_nearest = static_cast<int>(::round(ix));
        int iy_nearest = static_cast<int>(::round(iy));
        int iz_nearest = static_cast<int>(::round(iz));

        // assign nearest neighor pixel value to output pixel
        scalar_t *gOut_ptr_NCDHW = grad_output.data + n * gOut_sN + d * gOut_sD + h * gOut_sH + w * gOut_sW;
        scalar_t *gInp_ptr_NC = grad_input.data + n * gInp_sN;
        for (int c = 0; c < C; ++c, gOut_ptr_NCDHW += gOut_sC, gInp_ptr_NC += gInp_sC) {
          // calculate and set grad_input
          safe_add_3d(gInp_ptr_NC, iz_nearest, iy_nearest, ix_nearest,
                      gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, *gOut_ptr_NCDHW);
        }

        // assuming grad_grid is contiguous
        // thus we can
        //   1. use index with gGrid_sW to directly compute gGrid_ptr_NDHW
        //   2. directly assign to gGrid_ptr_NDHW[0], gGrid_ptr_NDHW[1], gGrid_ptr_NDHW[2]
        scalar_t *gGrid_ptr_NDHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NDHW[0] = static_cast<scalar_t>(0);
        gGrid_ptr_NDHW[1] = static_cast<scalar_t>(0);
        gGrid_ptr_NDHW[2] = static_cast<scalar_t>(0);
      }
    }
  }
}  // namespace

using namespace at;
// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
Tensor grid_sampler_2d_forward_cuda(const Tensor& input, const Tensor& grid,
                            int64_t interpolation_mode, int64_t padding_mode,
                            bool align_corners) {
  auto N = input.size(0);
  auto H = grid.size(1);
  auto W = grid.size(2);
  auto output = at::empty({N, input.size(1), H, W}, input.options());
  int count = static_cast<int>(N * H * W);
  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "grid_sampler_2d_forward_cuda", [&] {
      grid_sampler_2d_forward_kernel_cuda<scalar_t>
        <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
          count,
          getTensorInfo<scalar_t, int>(input),
          getTensorInfo<scalar_t, int>(grid),
          getTensorInfo<scalar_t, int>(output),
          static_cast<GridSamplerInterpolation>(interpolation_mode),
          static_cast<GridSamplerPadding>(padding_mode),
          align_corners);
    });
  }
  return output;
}

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
Tensor grid_sampler_3d_forward_cuda(const Tensor& input, const Tensor& grid,
                            int64_t interpolation_mode, int64_t padding_mode,
                            bool align_corners) {
  auto N = input.size(0);
  auto D = grid.size(1);
  auto H = grid.size(2);
  auto W = grid.size(3);
  auto output = at::empty({N, input.size(1), D, H, W}, input.options());
  int count = static_cast<int>(N * D * H * W);
  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "grid_sampler_3d_forward_cuda", [&] {
      grid_sampler_3d_forward_kernel_cuda<scalar_t>
        <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
          count,
          getTensorInfo<scalar_t, int>(input),
          getTensorInfo<scalar_t, int>(grid),
          getTensorInfo<scalar_t, int>(output),
          static_cast<GridSamplerInterpolation>(interpolation_mode),
          static_cast<GridSamplerPadding>(padding_mode),
          align_corners);
    });
  }
  return output;
}

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
std::tuple<Tensor, Tensor>
grid_sampler_2d_backward_cuda(const Tensor& grad_output, const Tensor& input,
                              const Tensor& grid, int64_t interpolation_mode,
                              int64_t padding_mode, bool align_corners) {
  auto N = input.size(0);
  auto H = grid.size(1);
  auto W = grid.size(2);
  auto grad_input = at::zeros_like(input);
  auto grad_grid = at::empty_like(grid);
  int count = static_cast<int>(N * H * W);
  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "grid_sampler_2d_backward_cuda", [&] {
      grid_sampler_2d_backward_kernel_cuda<scalar_t>
        <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
          count,
          getTensorInfo<scalar_t, int>(grad_output),
          getTensorInfo<scalar_t, int>(input),
          getTensorInfo<scalar_t, int>(grid),
          getTensorInfo<scalar_t, int>(grad_input),
          getTensorInfo<scalar_t, int>(grad_grid),
          static_cast<GridSamplerInterpolation>(interpolation_mode),
          static_cast<GridSamplerPadding>(padding_mode),
          align_corners);
    });
  }
  return std::make_tuple(grad_input, grad_grid);
}

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
std::tuple<Tensor, Tensor>
grid_sampler_3d_backward_cuda(const Tensor& grad_output, const Tensor& input,
                              const Tensor& grid, int64_t interpolation_mode, int64_t padding_mode,
                              bool align_corners) {
  auto N = input.size(0);
  auto D = grid.size(1);
  auto H = grid.size(2);
  auto W = grid.size(3);
  auto grad_input = at::zeros_like(input);
  auto grad_grid = at::empty_like(grid);
  int count = static_cast<int>(N * D * H * W);
  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "grid_sampler_3d_backward_cuda", [&] {
      grid_sampler_3d_backward_kernel_cuda<scalar_t>
        <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
          count,
          getTensorInfo<scalar_t, int>(grad_output),
          getTensorInfo<scalar_t, int>(input),
          getTensorInfo<scalar_t, int>(grid),
          getTensorInfo<scalar_t, int>(grad_input),
          getTensorInfo<scalar_t, int>(grad_grid),
          static_cast<GridSamplerInterpolation>(interpolation_mode),
          static_cast<GridSamplerPadding>(padding_mode),
          align_corners);
    });
  }
  return std::make_tuple(grad_input, grad_grid);
}

}  // namespace mmdetection
